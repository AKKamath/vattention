#include "hip/hip_runtime.h"
#include <stdio.h>
#include "vattn.h"

#define THREADS 1
#define BLOCKS 1

#define TEST_VAL    10
#define INDEX_2     (VATTN_DEF_HANDLE_SIZE / sizeof(int))
#define ARRAY_SIZE  (VATTN_MEM_RESERVE_LOW_LIMIT / sizeof(int))

int host[ARRAY_SIZE];

void __global__ test(int *a) {
    a[0] = TEST_VAL;
}

void __global__ test_2(int *a, int val) {
    // this kernel must be called after test.
    // this if case ensures that mapping indeed has been changed!
    if (a[0] != TEST_VAL)
        a[0] = val;
}

int main() {
    /* setup environment */
    if (vattn_init(0, VATTN_DEF_HANDLE_SIZE) != VATTN_OK) {
        printf("ERR!\n");
        exit(-1);
    }

    /* reserve memory */
    int *d = (int*)vattn_reserve_memory(VATTN_MEM_RESERVE_LOW_LIMIT);

    /* get_mem_handle */
    NvU64 handle_1;
    vattn_get_mem_handle(&handle_1);
    printf("HANDLE 1: %llx\n", handle_1);

    /* map handle to first 64k */
    vattn_mem_map((void*)d, handle_1);
    // Start the test
    test<<<1,1>>>(d);
    hipMemcpy(host, d, sizeof(int), hipMemcpyDeviceToHost);
    if (host[0] == TEST_VAL) {
        printf("YAY\n");
    }

    NvU64 handle_2;
    vattn_get_mem_handle(&handle_2);
    printf("HANDLE 2: %llx\n", handle_2);
    /* map new handle to the same location */
    vattn_mem_map((void*)d, handle_2);
    // Start the second test
    test_2<<<1,1>>>(d, TEST_VAL + 10);
    hipMemcpy(host, d, sizeof(int), hipMemcpyDeviceToHost);

    if (host[0] == TEST_VAL + 10) {
        printf("YAY\n");
    }

    vattn_free_reserved_address((void*)d, VATTN_MEM_RESERVE_LOW_LIMIT);
    vattn_release_mem_handle(handle_1);
    vattn_release_mem_handle(handle_2);
}
