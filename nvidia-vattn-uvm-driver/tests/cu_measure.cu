#include <iostream>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define ALLOC_SIZE                (2ULL * 1024 * 1024)
#define MAPPING_SIZE              (256ULL * 1024)
#define REQUESTS                  100                      // num of requests
#define LAYERS                    32                       // num of layers
#define TYPE                      2                        // KV

using namespace std;

hipCtx_t ctx;
hipMemAllocationProp prop = {};
hipMemAccessDesc accessDesc = {};

hipMemGenericAllocationHandle_t *handles;
hipDeviceptr_t *buffers;
size_t num_handles;
size_t block_size;

#define CHECK_CUDA(x) \
    do { \
        hipError_t res = x; \
        if (res != hipSuccess) { \
            const char *errStr = NULL; \
            (void)hipDrvGetErrorString(res, &errStr); \
            std::cerr << __FILE__ << ':' << __LINE__ << ' ' << #x \
                      << "failed (" << (unsigned)res << "): " << errStr << std::endl; \
        } \
    } while (0)

#define PROFILE_CALL(x, show) \
    do { \
        auto start = std::chrono::high_resolution_clock::now(); \
        x; \
        auto end = std::chrono::high_resolution_clock::now(); \
        if (show) \
            std::cout << #x << " took " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << "us" << std::endl; \
    } while (0)

static void do_cuda_init() {
    CHECK_CUDA(hipInit(0));
    CHECK_CUDA(hipCtxCreate(&ctx, 0, 0));
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = 0;
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    accessDesc.location.id = 0;
    CHECK_CUDA(hipMemGetAllocationGranularity(&block_size, &prop, hipMemAllocationGranularityMinimum));
}

void reserve_memory() {
    buffers = (hipDeviceptr_t*)malloc(num_handles * sizeof(hipDeviceptr_t));
    for (int i = 0; i < num_handles; i++) {
        hipMemAddressReserve(&buffers[i], block_size, 0, 0, 0);
    }
}

void get_handles() {
    handles = (hipMemGenericAllocationHandle_t*)malloc(num_handles * sizeof(hipMemGenericAllocationHandle_t));
    for (int i = 0; i < num_handles; i++) {
        hipMemCreate(&handles[i], block_size, &prop, 0);
    }
}

void map_handles() {
    for (int i = 0; i < num_handles; i++) {
        hipMemMap(buffers[i], block_size, 0, handles[i], 0);
        hipMemSetAccess(buffers[i], block_size, &accessDesc, 1);
    }
}

void free_buffers() {
    for (int i = 0; i < num_handles; i++) {
        hipMemUnmap(buffers[i], block_size);
        hipMemAddressFree(buffers[i], block_size);
    }
    free(buffers);
}

void release_handles() {
    for (int i = 0; i < num_handles; i++) {
        hipMemRelease(handles[i]);
    }
    free(handles);
}

void print_config() {
    printf("Number of handles: %lu\n", num_handles);
    printf("Number of requests: %d\n", REQUESTS);
    printf("Number of layers: %d\n", LAYERS);
    printf("Number of types: %d\n", TYPE);
    printf("Mapping size: %lu B\n", block_size);
    printf("Block size: %lu\n", block_size);
    printf("Total memory mapped: %lu MB\n", (num_handles * block_size) / (1024 * 1024));
}

int main(int argc, char **argv) {
    /* setup environment */
    do_cuda_init();

    // calculate the number of handles needed per token, per layer, per request for each KV
    num_handles = MAPPING_SIZE / block_size;
    if (num_handles <= 0)
        // block size > MAPPING_SIZE
        num_handles = 1;
    // calculate number of handles needed in total
    num_handles = (num_handles * TYPE * REQUESTS * LAYERS);

    print_config();
    PROFILE_CALL(reserve_memory(), false);
    PROFILE_CALL(get_handles(), false);
    PROFILE_CALL(map_handles(), true);

    PROFILE_CALL(free_buffers(), false);
    PROFILE_CALL(release_handles(), false);
    return 0;
}
